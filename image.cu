#include "hip/hip_runtime.h"
#include <image.h>

Image::Defaults Image::Default;

// Lab color space
__constant__ __device__ float Lab_M[9];
__constant__ __device__ float Lab_Mi[9];
__constant__ __device__ float3 Lab_W;

#define Lab_e 0.008856f
#define Lab_k 903.3f
#define Lab_v 0.0031308f
#define Lab_vi 0.04045f

static void setup_cielab(hipStream_t stream)
{
	static bool isInitialized = false;
	if (isInitialized) return;
	isInitialized = true;

	int rc;
        float pW[3] = { 0.95047f, 1.0f, 1.08883f };
        float pM[9] = {
                 0.4124f, 0.3576f, 0.1805f,
                 0.2126f, 0.7152f, 0.0722f,
                 0.0193f, 0.1192f, 0.9504f,
        };
        float pMi[9] = {
                 3.2406f,-1.5372f,-0.4986f,
                -0.9689f, 1.8758f, 0.0415f,
                 0.0557f, -0.2040, 1.0571f,
        };
        rc = hipMemcpyToSymbolAsync(HIP_SYMBOL(Lab_M), &pM, 9*sizeof(float), 0, hipMemcpyHostToDevice,stream);
        if (hipSuccess != rc) throw "Unable to copy cielab chromacity matrix";

        rc = hipMemcpyToSymbolAsync(HIP_SYMBOL(Lab_Mi), &pMi, 9*sizeof(float), 0, hipMemcpyHostToDevice,stream);
        if (hipSuccess != rc) throw "Unable to copy cielab inverted chromacity matrix";

        rc = hipMemcpyToSymbolAsync(HIP_SYMBOL(Lab_W), &pW, sizeof(float3), 0, hipMemcpyHostToDevice, stream);
        if (hipSuccess != rc) throw "Unable to copy cielab reference white"; 
}

__global__
void f_srgb_to_device(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        auto s = View<float4>(in, pitch_in, x, y);
	float4 d = make_float4(
			s.x < 0.04045f ? s.x/12.92f : pow((s.x + 0.055f)/1.055f, 2.4f),
			s.y < 0.04045f ? s.y/12.92f : pow((s.y + 0.055f)/1.055f, 2.4f),
			s.z < 0.04045f ? s.z/12.92f : pow((s.z + 0.055f)/1.055f, 2.4f),
			s.w);

	View<float4>(out, pitch_out, x, y) = d;
}
__global__
void f_device_to_srgb(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        auto s = View<float4>(in, pitch_in, x, y);
	float4 d = make_float4(
			s.x < 0.0031308f ? s.x*12.92f : pow(1.055f * s.x, 1.0f/2.4f)-0.055,
			s.y < 0.0031308f ? s.y*12.92f : pow(1.055f * s.y, 1.0f/2.4f)-0.055,
			s.z < 0.0031308f ? s.z*12.92f : pow(1.055f * s.z, 1.0f/2.4f)-0.055,
			s.w);

	View<float4>(out, pitch_out, x, y) = d;
}

__global__
void f_device_to_grayscale(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        auto s = View<float4>(in, pitch_in, x, y);
	float l = 0.2126f * s.x + 0.7152f * s.y + 0.0722f * s.z;
	View<float4>(out, pitch_out, x, y) = make_float4(l, l, l, s.w);
}

__global__
void f_srgb_to_grayscale(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        auto s = View<float4>(in, pitch_in, x, y);
	float4 d = make_float4(
			s.x < 0.04045f ? s.x/12.92f : pow((s.x + 0.055f)/1.055f, 2.4f),
			s.y < 0.04045f ? s.y/12.92f : pow((s.y + 0.055f)/1.055f, 2.4f),
			s.z < 0.04045f ? s.z/12.92f : pow((s.z + 0.055f)/1.055f, 2.4f),
			s.w);
	float l = 0.2126f * d.x + 0.7152f * d.y + 0.0722f * d.z;

	View<float4>(out, pitch_out, x, y) = make_float4(l,l,l,s.w);
}

__global__
void f_grayscale_to_srgb(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        auto s = View<float4>(in, pitch_in, x, y);
	float d = s.x < 0.0031308f ? s.x*12.92f : pow(1.055f * s.x, 1.0f/2.4f)-0.055;
	View<float4>(out, pitch_out, x, y) = make_float4(d,d,d, s.w);
}
__global__
void f_srgb_to_cielab(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        auto p = View<float4>(in, pitch_in, x, y);
        auto RGB = make_float3(p.x, p.y, p.z);

        float3 rgb = make_float3(
                RGB.x <= Lab_vi ? RGB.x / 12.92 : pow((RGB.x + 0.055)/1.055, 2.4),
                RGB.y <= Lab_vi ? RGB.y / 12.92 : pow((RGB.y + 0.055)/1.055, 2.4),
                RGB.z <= Lab_vi ? RGB.z / 12.92 : pow((RGB.z + 0.055)/1.055, 2.4)
        );

        float3 xyz = make_float3(
                Lab_M[0] * rgb.x + Lab_M[1] * rgb.y + Lab_M[2] * rgb.z,
                Lab_M[3] * rgb.x + Lab_M[4] * rgb.y + Lab_M[5] * rgb.z,
                Lab_M[6] * rgb.x + Lab_M[7] * rgb.y + Lab_M[8] * rgb.z
        );
        float3 r = make_float3(
                xyz.x / Lab_W.x,
                xyz.y / Lab_W.y,
                xyz.z / Lab_W.z
        );
        float3 f = make_float3(
                r.x > Lab_e ? pow(r.x, 1.0f/3.0f) : (Lab_k * r.x + 16.0f) / 116.0f,
                r.y > Lab_e ? pow(r.y, 1.0f/3.0f) : (Lab_k * r.y + 16.0f) / 116.0f,
                r.z > Lab_e ? pow(r.z, 1.0f/3.0f) : (Lab_k * r.z + 16.0f) / 116.0f
        );

        float4 Lab = make_float4(
                        116.0f * f.y - 16.0f,
                        500.0f * (f.x - f.y),
                        200.0f * (f.y - f.z),
			p.w);

        View<float4>(out, pitch_out, x, y) = Lab;
}

__global__
void f_cielab_to_srgb(float4* out, size_t pitch_out, float4* in, size_t pitch_in, size_t width, size_t height)
{
        int x = (blockIdx.x * blockDim.x + threadIdx.x);
        int y = (blockIdx.y * blockDim.y + threadIdx.y);
        if (x >= width || y >= height) return;

        float4 Lab = View<float4>(in, pitch_in, x, y);
        float3 f = make_float3(
                (Lab.x+16.0f)/116.0f + Lab.y/500.0f,
                (Lab.x+16.0f)/116.0f,
                (Lab.x+16.0f)/116.0f - Lab.z/200.0f
        );
        float3 f3 = make_float3(
                f.x * f.x * f.x,
                f.y * f.y * f.y,
                f.z * f.z * f.z
        );
        float3 r = make_float3(
                f3.x > Lab_e ? f3.x : (116.0f * f.x - 16.0f)/Lab_k,
                f3.y > Lab_e ? f3.y : (116.0f * f.y - 16.0f)/Lab_k,
                f3.z > Lab_e ? f3.z : (116.0f * f.z - 16.0f)/Lab_k
                //Lab.x > Lab_k*Lab_e ? f3.y : Lab.x/Lab_k
        );
        float3 xyz = make_float3(
                r.x * Lab_W.x,
                r.y * Lab_W.y,
                r.z * Lab_W.z
        );
        float3 rgb = make_float3(
                Lab_Mi[0] * xyz.x + Lab_Mi[1] * xyz.y + Lab_Mi[2] * xyz.z,
                Lab_Mi[3] * xyz.x + Lab_Mi[4] * xyz.y + Lab_Mi[5] * xyz.z,
                Lab_Mi[6] * xyz.x + Lab_Mi[7] * xyz.y + Lab_Mi[8] * xyz.z
        );
        float3 RGB = make_float3(
                rgb.x <= Lab_v ? 12.92f * rgb.x : 1.055f * pow(rgb.x, 1.0f/2.4f) - 0.055,
                rgb.y <= Lab_v ? 12.92f * rgb.y : 1.055f * pow(rgb.y, 1.0f/2.4f) - 0.055,
                rgb.z <= Lab_v ? 12.92f * rgb.z : 1.055f * pow(rgb.z, 1.0f/2.4f) - 0.055
        );
        View<float4>(out, pitch_out, x, y) = 
		make_float4(
			clamp(RGB.x, 0.0f, 1.0f), 
			clamp(RGB.y, 0.0f, 1.0f),
			clamp(RGB.z, 0.0f, 1.0f),
			Lab.w
		);
}


static const char* strrstr(const char* c, const char* find)
{
	if (!c || !find) return nullptr;
	if (strlen(find) == 0) return c + strlen(c);
	if (strlen(c) < strlen(find)) return nullptr;

	for (int i=strlen(c)-strlen(find); i >= 0; i--)
	{
		if (!memcmp(c + i, find, strlen(find))) return c + i;
	}
	return nullptr;
}

Image::~Image()
{
	if (_filename) free(_filename);
	if (mem.host.data) hipHostFree(mem.host.data);
	if (mem.device.data) hipFree(mem.device.data);
}

void Image::toHost(hipStream_t stream)
{
	int rc = hipMemcpy2DAsync(
			mem.host.data, mem.host.pitch, 
			mem.device.data, mem.device.pitch, 
			width * sizeof(float4), 
			height,
			hipMemcpyDeviceToHost, 
			stream); 
	if (hipSuccess != rc) throw "Unable to copy from device to host";
}
void Image::toDevice(hipStream_t stream)
{
	int rc = hipMemcpy2DAsync(
			mem.device.data, mem.device.pitch, 
			mem.host.data, mem.host.pitch, 
			width * sizeof(float4), 
			height,
			hipMemcpyHostToDevice, 
			stream); 
	if (hipSuccess != rc) throw "Unable to copy from host to device";
}

#define FROM_TO(a, b) ((int)(a)<<8|(int)(b))

void Image::convert(ColorSpace cs, hipStream_t stream)
{
	if (cs == ColorSpace::Default) cs = Image::Default.colorSpace;
	setup_cielab(stream);
	
	dim3 blockSize = { 16, 16 };
	dim3 gridSize = {
		((int)width  + blockSize.x - 1) / blockSize.x,
		((int)height + blockSize.y - 1) / blockSize.y
	};

	switch (FROM_TO(colorSpace, cs))
	{
		case FROM_TO(ColorSpace::sRGB, ColorSpace::Device):
			f_srgb_to_device <<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;

		case FROM_TO(ColorSpace::Device, ColorSpace::sRGB):
			f_device_to_srgb <<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;
		case FROM_TO(ColorSpace::sRGB, ColorSpace::CIELab):
			f_srgb_to_cielab <<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;

		case FROM_TO(ColorSpace::CIELab, ColorSpace::sRGB):
			f_cielab_to_srgb <<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;
		
		case FROM_TO(ColorSpace::Grayscale, ColorSpace::sRGB):
			f_grayscale_to_srgb <<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;
		
		case FROM_TO(ColorSpace::sRGB, ColorSpace::Grayscale):
			f_srgb_to_grayscale<<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;
		
		case FROM_TO(ColorSpace::Device, ColorSpace::Grayscale):
			f_device_to_grayscale<<< gridSize, blockSize, 0, stream >>> (
				mem.device.data, mem.device.pitch,
				mem.device.data, mem.device.pitch,
				width, height);
			break;
		
		case FROM_TO(ColorSpace::Grayscale, ColorSpace::Device):
			//nop
			break;

		default: 
			throw "Conversion not implemented yet";
	}
	colorSpace = cs;
}

void Image::printInfo()
{
	printf("IMAGE %s\n", filename);
	printf("- SIZE:  %lu x %lu\n", width, height);
	printf("- PITCH: %lu (dev), %lu (host)\n", mem.device.pitch, mem.host.pitch);
	printf("\n");
	fflush(stdout);
}

void Image::alloc(size_t w, size_t h, size_t c)
{
	int rc;
	if (mem.host.data) hipHostFree(mem.host.data);
	if (mem.device.data) hipFree(mem.device.data);

	width = w;
	height = h;
	channels = c;

	mem.host.pitch = width * sizeof(float4);
	rc = hipHostMalloc(&mem.host.data, mem.host.pitch * height);
	if (hipSuccess != rc) throw "Unable to allocate host memory for image";

	rc = hipMallocPitch(&mem.device.data, &mem.device.pitch, 
			width * sizeof(float4), height);
	if (hipSuccess != rc) throw "Unable to allocate device memory for image";
}

void Image::loadPPM()
{
	int rc;

	FILE* f = fopen(filename, "rb");
	if (!f) throw "Unable to open file";

	size_t range;
	rc = fscanf(f, "P6 %lu %lu %lu \n", &width, &height, &range);
	if (rc <= 0) throw "Unable to read PPM header";

	int bpp;
	channels = 3;
	if (range < 256) bpp = 8;
	else bpp = 16;

	mem.host.pitch = width * sizeof(float4);
	size_t pitch = width * (bpp >> 3) * channels;
	rc = hipHostMalloc(&mem.host.data, width * sizeof(float4) * height);
	if (hipSuccess != rc) throw "Unable to allocate host memory for image";

	rc = hipMallocPitch(&mem.device.data, &mem.device.pitch, width * sizeof(float4), height);
	if (hipSuccess != rc) throw "Unable to allocate device memory for image";

	rc = fread(mem.host.data, 1, pitch * height, f);
	if (rc <= 0) throw "Unable to read image data from PPM";

	for (int i= width * height - 1; i>= 0; i--)
	{
		if (bpp == 8)
		{
			uchar3 p = ((uchar3*)mem.host.data)[i];
			mem.host.data[i] = make_float4(p.x, p.y, p.z, range) / (float)range;
		}
		else
		{
			ushort3 p = ((ushort3*)mem.host.data)[i];
			mem.host.data[i] = make_float4(p.x, p.y, p.z, range) / (float)range;
		}
	}
	fclose(f);
	toDevice();
}

void Image::loadPGM()
{
	throw "Not implemented yet";
}

void Image::loadJPG()
{
	throw "Not implemented yet";
}

void Image::load(const char* path)
{
	filename = _filename = strdup(path);

	const char* extension = strrstr(filename, ".");

	if (!strcmp(extension, ".jpeg")) loadJPG();
	if (!strcmp(extension, ".jpg"))  loadJPG();
	if (!strcmp(extension, ".ppm"))  loadPPM();
	if (!strcmp(extension, ".pgm"))  loadPGM();
}

float Image::psnr(const Image* ref)
{
	float mse = 0;
	for (size_t x=2; x<width-2; x++)
	{
		for (size_t y=2; y<height-2; y++)
		{
			for (size_t c=0; c<channels; c++)
			{
				void* p = ((uint8_t*)mem.host.data
						+ y * mem.host.pitch
						+ (x * sizeof(float4)));

				void* q = ((uint8_t*)ref->mem.host.data
						+ y * ref->mem.host.pitch
						+ (x * sizeof(float4)));
				float pv = ((float*)p)[c];
				float qv = ((float*)q)[c];

				mse += (pv - qv) * (pv - qv);
			}
		}
	}
	mse /= width * height * channels;
	return 20 * log10(1.0) - 10 * log10(mse);
}

JpegCodec::JpegCodec()
{
	_width = 0;
	_height = 0;
	_channels = 0;
	_buffer = nullptr;
	_scanlines = nullptr;
	
	_dinfo.err = jpeg_std_error(&_djerr);
	_cinfo.err = jpeg_std_error(&_cjerr);
}
	
JpegCodec::~JpegCodec()
{
	free(_buffer);
	free(_scanlines);
}

void JpegCodec::prepare(int width, int height, int channels, int quality)
{
	if (channels != 3) throw "Not implemented channels != 3";

	_width = width;
	_height = height;
	_channels = channels;

	_buffer = (uint8_t*) malloc(_width * _height * _channels);
	if (!_buffer) throw "Unable to allocate intermediate buffer";

	_scanlines = (JSAMPARRAY) malloc( sizeof(JSAMPROW) * height);
	if (!_scanlines)
	{
		free(_buffer);
		throw "Unable to allocate scanlines structure";
	}

	for (size_t i=0; i<_height; i++)
	{
		_scanlines[i] = (JSAMPROW) (_buffer + i * _width * _channels);
	}

	jpeg_create_decompress(&_dinfo);
	jpeg_create_compress(&_cinfo);
	
	_cinfo.image_width = _width;
	_cinfo.image_height = height;
	_cinfo.input_components = 3;
	_cinfo.in_color_space = JCS_RGB; 
	jpeg_set_defaults(&_cinfo);
	jpeg_set_quality(&_cinfo, quality, 1);
}

void JpegCodec::unprepare()
{
	jpeg_destroy_decompress(&_dinfo);
	jpeg_destroy_compress(&_cinfo);
}

void JpegCodec::encodeCPU(void* dst, size_t *size)
{
	//hipMemcpyAsync(_buffer, src, _width * _height * _channels, hipMemcpyDeviceToHost, stream);
	//hipStreamSynchronize(stream);
	
	jpeg_mem_dest(&_cinfo, (uint8_t**)&dst, size);
	jpeg_start_compress(&_cinfo, 1);
	while (_cinfo.next_scanline < _cinfo.image_height)
	{
		jpeg_write_scanlines(&_cinfo, _scanlines + _cinfo.next_scanline, _cinfo.image_height - _cinfo.next_scanline);
	}
	jpeg_finish_compress(&_cinfo);
}

void JpegCodec::decodeToDeviceMemoryCPU(void* dst, const void* src, size_t size, hipStream_t stream)
{
	jpeg_mem_src(&_dinfo, (uint8_t*)src, size);
	jpeg_read_header(&_dinfo, 1);
	jpeg_calc_output_dimensions(&_dinfo);

	if (_dinfo.output_width != _width 
	||  _dinfo.output_height != _height
	||  _dinfo.output_components != (int) _channels)
	{
		jpeg_abort_decompress(&_dinfo);
		throw "Invalid image format";
	}
	jpeg_start_decompress(&_dinfo);
	while (_dinfo.output_scanline < _dinfo.output_height)
	{
		jpeg_read_scanlines(&_dinfo, _scanlines + _dinfo.output_scanline,_dinfo.output_height - _dinfo.output_scanline);
	}
	jpeg_finish_decompress(&_dinfo);

	hipMemcpyAsync(dst, _buffer, _width * _height * _channels, hipMemcpyHostToDevice, stream);
}

#if USE_NVJPEG
void JpegCodec::decodeToDeviceMemoryGPU(void* dst, const void* src, size_t size, hipStream_t stream)
{
	int rc;
	
	nvjpegHandle_t handle;
	rc = nvjpegCreateEx(NVJPEG_BACKEND_DEFAULT, NULL, NULL, 0, &handle);
	if (hipSuccess != rc) throw "Unable to create nvjpeg handle";

	int channels;
	int widths[NVJPEG_MAX_COMPONENT];
	int heights[NVJPEG_MAX_COMPONENT];
	nvjpegChromaSubsampling_t subsampling;
	nvjpegJpegState_t state;
	nvjpegOutputFormat_t fmt = NVJPEG_OUTPUT_RGBI;
	nvjpegJpegStateCreate(handle, &state);
	nvjpegGetImageInfo(handle, (uint8_t*) src, size, &channels, &subsampling, widths, heights);

	if (widths[0] != (int)_width
	||  heights[0] != (int)_height)
	{
		nvjpegJpegStateDestroy(state);
		nvjpegDestroy(handle);
		throw "Invalid image format";
	}

	nvjpegImage_t output;
	output.channel[0] = (uint8_t*) dst;
	output.pitch[0] = widths[0] * _channels;

	nvjpegDecode(handle, state, (uint8_t*)src, size, fmt, &output, stream);
	nvjpegJpegStateDestroy(state);
	nvjpegDestroy(handle);

}
#endif
